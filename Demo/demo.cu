#include "hip/hip_runtime.h"
#include <stdio.h>
#include "headfile.h"
#include "kernels.h"


#define _patch_size 16;
#define _patch_length 1089;//(2*16+1)*(2*16+1)
#define _image_height 375;
#define _image_width 450;

__constant__ AlogrithConfigFloat dev_algorith_const;
__constant__ int dev_aggr_const[1024];
__constant__ int3 dev_aggr_const3[1024];
__constant__ int dev_patch_size = _patch_size;
__constant__ int kWidthDev = 450;
__constant__ int kHeightDev = 375;
__constant__ unsigned int kWidthAddDev = 451;
__constant__ unsigned int kHeightAddDev = 376;
__constant__ unsigned int kWidthSubDev = 449;
__constant__ unsigned int kHeightSubDev = 374;

__constant__ float kSoblexDev[9] = { -1,0,1,-2,0,3,-1,0,1 };
__constant__ float kSobleyDev[9] = { 1,2,1,0,0,0,-1,-2,-1 };



#define DEBUG 1

#ifdef DEBUG
	#include "opencv2/opencv.hpp"
#endif // DEBUG


#define checkCudaErrors(val) check((val), #val, __FILE__, __LINE__)
template <typename T>
void check(T result, char const* const func, const char* const file, int const line) {
	if (result) {
		fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\" \n", file, line,
			static_cast<unsigned int>(result), hipGetErrorName(result), func);
		exit(EXIT_FAILURE);
	}
}



int main()
{
	hipError_t runtime_status = hipSuccess;
	int patch_size = _patch_size;
	int patch_length = _patch_length;
	int id = 0;
	int *host_aggr_offset = NULL;
	hipHostMalloc((void**)&host_aggr_offset, sizeof(int) * patch_length);
	int3 *host_aggr3_offset = NULL;
	hipHostMalloc((void**)&host_aggr3_offset, sizeof(int3) * patch_length);

	for (int row = -patch_size; row < patch_size; ++row)
	{
		for (int col = -patch_size; col < patch_size; ++col)
		{
			host_aggr_offset[id] = row * _image_width + col;
			host_aggr3_offset[id] = make_int3(col, row, host_aggr_offset[id]);
			++id;
		}
	}
	AlogrithConfigFloat host_algorith;
	host_algorith.alpha = 0.9;
	host_algorith.cost_punish = 120;
	host_algorith.disparity_range = 64;
	host_algorith.gamma = 0.1;
	host_algorith.is_check_lr = 1;
	host_algorith.is_fill_holes = 1;
	host_algorith.is_fource_fpw = 1;
	host_algorith.is_integer_disp = 1;
	host_algorith.lrcheck_thres = 10;
	host_algorith.max_disparity = 64;
	host_algorith.min_disparity = 0;
	host_algorith.tau_color = 10;
	host_algorith.tau_grad = 2;



	runtime_status = InitConstParams(&host_algorith, host_aggr3_offset, host_aggr_offset);
	checkCudaErrors(runtime_status);

	hipDeviceProp_t devprop;
	runtime_status = hipGetDeviceProperties(&devprop, 0);
	checkCudaErrors(runtime_status);
	size_t maxblockperSM = devprop.maxBlocksPerMultiProcessor;
	size_t maxthreadperBlock = devprop.maxThreadsPerBlock;
	size_t maxthreadperSM = devprop.maxThreadsPerMultiProcessor;
	size_t maxregisterperSM = devprop.regsPerMultiprocessor;
	size_t maxregisterperblock = devprop.regsPerBlock;
	size_t num_SM = devprop.multiProcessorCount;

	hiprandStatus_t cuda_rand_status = HIPRAND_STATUS_SUCCESS;
	ImageInfo bgrInfo(450u, 375u, Uchar3Img);
	ImageInfo grayInfo(450u, 375u, UcharImg);

	// memory allocate
	uchar *hostbgrleft = NULL;
	uchar *hostbgrright = NULL;

	float* hostgrayleft = NULL;
	float* hostgrayright = NULL;
	float3* hostgradleft = NULL;
	float3* hostgradright = NULL;

	float *hostdispleft = NULL;
	float *hostdispright = NULL;
	float3 *hostdispplaneleft = NULL;
	float3 *hostdispplaneright = NULL;
	float *hostcostaggrleft = NULL;
	float *hostcostaggrright = NULL;


	uchar3* devbgrleft = NULL;
	uchar3* devbgrright = NULL;
	float* devgrayleft = NULL;
	float* devgrayright = NULL;
	float3* devgradleft = NULL;
	float3* devgradright = NULL;


	float *devdispleft = NULL;
	float *devdispright = NULL;
	float3 *devdispplaneleft = NULL;
	float3 *devdispplaneright = NULL;
	float *devcostaggrleft = NULL;
	float *devcostaggrright = NULL;

	//host
	runtime_status = hipHostMalloc((void**)&hostbgrleft, sizeof(uchar) * grayInfo.imgsize * 3);
	checkCudaErrors(runtime_status);
	runtime_status = hipHostMalloc((void**)&hostbgrright, sizeof(uchar) * grayInfo.imgsize * 3);
	checkCudaErrors(runtime_status);
	runtime_status = hipHostMalloc((void**)&hostgrayleft, sizeof(float)*grayInfo.imgsize);
	checkCudaErrors(runtime_status);
	runtime_status = hipHostMalloc((void**)&hostgrayright, sizeof(float) * grayInfo.imgsize);
	checkCudaErrors(runtime_status);
	runtime_status = hipHostMalloc((void**)&hostgradleft, sizeof(float3) * grayInfo.imgsize);
	checkCudaErrors(runtime_status);
	runtime_status = hipHostMalloc((void**)&hostgradright, sizeof(float3) * grayInfo.imgsize);
	checkCudaErrors(runtime_status);
	runtime_status = hipHostMalloc((void**)&hostdispleft, sizeof(float) * grayInfo.imgsize);
	checkCudaErrors(runtime_status);
	runtime_status = hipHostMalloc((void**)&hostdispright, sizeof(float) * grayInfo.imgsize);
	checkCudaErrors(runtime_status);
	runtime_status = hipHostMalloc((void**)&hostdispplaneleft, sizeof(float3) * grayInfo.imgsize);
	checkCudaErrors(runtime_status);
	runtime_status = hipHostMalloc((void**)&hostdispplaneright, sizeof(float3) * grayInfo.imgsize);
	checkCudaErrors(runtime_status);

	runtime_status = hipHostMalloc((void**)&hostcostaggrleft, sizeof(float) * grayInfo.imgsize);
	checkCudaErrors(runtime_status);
	runtime_status = hipHostMalloc((void**)&hostcostaggrright, sizeof(float) * grayInfo.imgsize);
	checkCudaErrors(runtime_status);

	//gray image and grad image
	runtime_status = hipMalloc((void**)&devgrayleft, sizeof(float) * grayInfo.imgsize);
	checkCudaErrors(runtime_status);
	runtime_status = hipMalloc((void**)&devgrayright, sizeof(float) * grayInfo.imgsize);
	checkCudaErrors(runtime_status);
	runtime_status = hipMalloc((void**)&devbgrleft, sizeof(float) * grayInfo.imgsize);
	checkCudaErrors(runtime_status);
	runtime_status = hipMalloc((void**)&devbgrright, sizeof(float) * grayInfo.imgsize);
	checkCudaErrors(runtime_status);
	runtime_status = hipMalloc((void**)&devgradleft, sizeof(float3) * grayInfo.imgsize);
	checkCudaErrors(runtime_status);
	runtime_status = hipMalloc((void**)&devgradright, sizeof(float3) * grayInfo.imgsize);
	checkCudaErrors(runtime_status);

	//disparity image and disparity plane image and cost aggr image
	runtime_status = hipMalloc((void**)&devdispleft, sizeof(float) * grayInfo.imgsize);
	checkCudaErrors(runtime_status);
	runtime_status = hipMalloc((void**)&devdispright, sizeof(float) * grayInfo.imgsize);
	checkCudaErrors(runtime_status);
	runtime_status = hipMalloc((void**)&devdispplaneleft, sizeof(float3) * grayInfo.imgsize);
	checkCudaErrors(runtime_status);
	runtime_status = hipMalloc((void**)&devdispplaneright, sizeof(float3) * grayInfo.imgsize);
	checkCudaErrors(runtime_status);
	runtime_status = hipMalloc((void**)&devcostaggrleft, sizeof(float) * grayInfo.imgsize);
	checkCudaErrors(runtime_status);
	runtime_status = hipMalloc((void**)&devcostaggrright, sizeof(float) * grayInfo.imgsize);
	checkCudaErrors(runtime_status);

	//
	runtime_status = hipMemset(devdispleft, 0, sizeof(float) * grayInfo.imgsize);
	checkCudaErrors(runtime_status);
	runtime_status = hipMemset(devdispright, 0, sizeof(float) * grayInfo.imgsize);
	checkCudaErrors(runtime_status);
	runtime_status = hipMemset(devdispplaneleft, 0, sizeof(float3) * grayInfo.imgsize);
	checkCudaErrors(runtime_status);
	runtime_status = hipMemset(devdispplaneright, 0, sizeof(float3) * grayInfo.imgsize);
	checkCudaErrors(runtime_status);
	runtime_status = hipMemset(devcostaggrleft, 0, sizeof(float) * grayInfo.imgsize);
	checkCudaErrors(runtime_status);
	runtime_status = hipMemset(devcostaggrright, 0, sizeof(float) * grayInfo.imgsize);
	checkCudaErrors(runtime_status);

	std::cout << sizeof(float3) << std::endl;


	// init device disparity plane
	hiprandGenerator_t generator;
	hiprandStatus_t rand_status = HIPRAND_STATUS_SUCCESS;
	rand_status = hiprandCreateGenerator(&generator, HIPRAND_RNG_QUASI_SCRAMBLED_SOBOL32);
	if (HIPRAND_STATUS_SUCCESS != rand_status)
	{
		return rand_status;
	}

	rand_status = hiprandGenerateUniform(generator, devdispleft, grayInfo.imgsize);
	checkCudaErrors(hipError_t(cuda_rand_status));
	rand_status = hiprandGenerateUniform(generator, devdispright, grayInfo.imgsize);
	checkCudaErrors(hipError_t(cuda_rand_status));	
	rand_status = hiprandGenerateUniform(generator, (float*)(void*)devdispplaneleft, grayInfo.imgsize*3);
	checkCudaErrors(hipError_t(cuda_rand_status));
	rand_status = hiprandGenerateUniform(generator, (float*)(void*)devdispplaneright,grayInfo.imgsize*3);
	checkCudaErrors(hipError_t(cuda_rand_status));

	#ifdef DEBUG

	runtime_status = hipMemcpy(hostdispleft, devdispleft, sizeof(float) * grayInfo.imgsize, hipMemcpyDeviceToHost);
	checkCudaErrors(runtime_status);
	runtime_status = hipMemcpy(hostdispright, devdispright, sizeof(float) * grayInfo.imgsize, hipMemcpyDeviceToHost);
	checkCudaErrors(runtime_status);
	runtime_status = hipMemcpy(hostdispplaneleft, devdispplaneleft, sizeof(float3) * grayInfo.imgsize, hipMemcpyDeviceToHost);
	checkCudaErrors(runtime_status);
	runtime_status = hipMemcpy(hostdispplaneright, devdispplaneright, sizeof(float3) * grayInfo.imgsize, hipMemcpyDeviceToHost);
	checkCudaErrors(runtime_status);
	hipDeviceSynchronize();

	cv::Mat hostdispleft_mat = cv::Mat(grayInfo.height, grayInfo.width, CV_32FC1, hostdispleft);
	cv::Mat hostdispright_mat = cv::Mat(grayInfo.height, grayInfo.width, CV_32FC1, hostdispright);
	cv::Mat hostdispplaneleft_mat = cv::Mat(grayInfo.height, grayInfo.width, CV_32FC3, hostdispplaneleft);
	cv::Mat hostdispplaneright_mat = cv::Mat(grayInfo.height, grayInfo.width, CV_32FC3, hostdispplaneright);
	#endif // DEBUG

	dim3 threadsperblock(32u, 16u);
	dim3 blockpergrid(15u, 24u);
	runtime_status = RandomInitialDisparityAndItsPlane(devdispleft, devdispplaneleft,
										host_algorith.min_disparity, host_algorith.disparity_range,
										grayInfo.width, grayInfo.height, blockpergrid, threadsperblock);
	checkCudaErrors(runtime_status);
	runtime_status = RandomInitialDisparityAndItsPlane(devdispright, devdispplaneright,
										host_algorith.min_disparity, host_algorith.disparity_range,
										grayInfo.width, grayInfo.height, blockpergrid, threadsperblock);
	checkCudaErrors(runtime_status);

	hipDeviceSynchronize();
#ifdef DEBUG
	runtime_status = hipMemcpy(hostdispleft, devdispleft, sizeof(float) * grayInfo.imgsize, hipMemcpyDeviceToHost);
	checkCudaErrors(runtime_status);
	runtime_status = hipMemcpy(hostdispright, devdispright, sizeof(float) * grayInfo.imgsize, hipMemcpyDeviceToHost);
	checkCudaErrors(runtime_status);
	runtime_status = hipMemcpy(hostdispplaneleft, devdispplaneleft, sizeof(float3) * grayInfo.imgsize, hipMemcpyDeviceToHost);
	checkCudaErrors(runtime_status);
	runtime_status = hipMemcpy(hostdispplaneright, devdispplaneright, sizeof(float3) * grayInfo.imgsize, hipMemcpyDeviceToHost);
	checkCudaErrors(runtime_status);
	hipDeviceSynchronize();
#endif // DEBUG


//------------------------------------major code--------------------------------------//
	//load image from host memory
	FILE *fptr = NULL;
	fptr = fopen("../../data/Cone/left.raw", "rb");
	fread(hostbgrleft, sizeof(uchar), bgrInfo.width * bgrInfo.height * 3, fptr);
	fclose(fptr);
	fptr = fopen("../../data/Cone/right.raw", "rb");
	fread(hostbgrright, sizeof(uchar), bgrInfo.width * bgrInfo.height * 3, fptr);
	fclose(fptr);

	#ifdef DEBUG
		cv::Mat host_bgr_left_mat = cv::Mat(grayInfo.height, grayInfo.width, CV_8UC3, hostbgrleft);
		cv::Mat host_bgr_right_mat = cv::Mat(grayInfo.height, grayInfo.width, CV_8UC3, hostbgrright);
	#endif //DEBUG


	//Copy image to device
	runtime_status = hipMemcpy(devbgrleft, hostbgrleft, sizeof(char3) * grayInfo.imgsize, hipMemcpyHostToDevice);
	checkCudaErrors(runtime_status);
	runtime_status = hipMemcpy(devbgrright, hostbgrright, sizeof(char3) * grayInfo.imgsize, hipMemcpyHostToDevice);
	checkCudaErrors(runtime_status);
	hipDeviceSynchronize();
	//convert bgr image to gray and grad image
	runtime_status = Bgr2Gray(devgrayleft, devbgrleft,grayInfo.width, grayInfo.height, blockpergrid, threadsperblock);
	checkCudaErrors(runtime_status);
	runtime_status = Bgr2Gray(devgrayright, devbgrright,grayInfo.width, grayInfo.height, blockpergrid, threadsperblock);
	checkCudaErrors(runtime_status);
	runtime_status = Gray2Sobel(devgradleft,devgrayleft,  grayInfo.width, grayInfo.height, blockpergrid, threadsperblock);
	checkCudaErrors(runtime_status);
	runtime_status = Gray2Sobel(devgradright,devgrayright, grayInfo.width, grayInfo.height, blockpergrid, threadsperblock);
	checkCudaErrors(runtime_status);
	hipDeviceSynchronize();
	//load image to host
#ifdef DEBUG
	runtime_status = hipMemcpy(hostgrayleft, devgrayleft, sizeof(float) * grayInfo.imgsize, hipMemcpyDeviceToHost);
	checkCudaErrors(runtime_status);
	runtime_status = hipMemcpy(hostgrayright, devgrayright, sizeof(float) * grayInfo.imgsize, hipMemcpyDeviceToHost);
	checkCudaErrors(runtime_status);
	runtime_status = hipMemcpy(hostgradleft, devgradleft, sizeof(float3) * grayInfo.imgsize, hipMemcpyDeviceToHost);
	checkCudaErrors(runtime_status);
	runtime_status = hipMemcpy(hostgradright, devgradright, sizeof(float3) * grayInfo.imgsize, hipMemcpyDeviceToHost);
	checkCudaErrors(runtime_status);
	hipDeviceSynchronize();

	cv::Mat gray_left_mat = cv::Mat(grayInfo.height, grayInfo.width, CV_32FC1, hostgrayleft);
	cv::Mat grad_left_mat = cv::Mat(grayInfo.height, grayInfo.width, CV_32FC3, hostgradleft);
	cv::Mat gray_right_mat = cv::Mat(grayInfo.height, grayInfo.width, CV_32FC1, hostgrayright);
	cv::Mat grad_right_mat = cv::Mat(grayInfo.height, grayInfo.width, CV_32FC3, hostgradright);
#endif //DEBUG


	runtime_status = CaculateCostAggregationInitConst(devcostaggrleft, devdispplaneleft,
								devbgrleft, devbgrright,
								devgradleft, devgradright,
								blockpergrid, threadsperblock);

	checkCudaErrors(runtime_status);

	runtime_status = CaculateCostAggregationInitConst(devcostaggrright, devdispplaneright,
								devbgrright, devbgrleft,
								devgradright, devgradleft,
								blockpergrid, threadsperblock);

	checkCudaErrors(runtime_status);
	hipDeviceSynchronize();

#ifdef DEBUG

	runtime_status = hipMemcpy(hostcostaggrleft, devcostaggrleft, sizeof(float) * grayInfo.imgsize, hipMemcpyDeviceToHost);
	checkCudaErrors(runtime_status);
	runtime_status = hipMemcpy(hostcostaggrright, devcostaggrright, sizeof(float) * grayInfo.imgsize, hipMemcpyDeviceToHost);
	checkCudaErrors(runtime_status);
	hipDeviceSynchronize();

	cv::Mat hostcostaggrleft_mat = cv::Mat(grayInfo.height, grayInfo.width, CV_32FC1, hostcostaggrleft);
	cv::Mat hostcostaggrright_mat = cv::Mat(grayInfo.height, grayInfo.width, CV_32FC1, hostcostaggrright);

#endif // DEBUG


	runtime_status = hipHostFree(hostbgrleft);
	checkCudaErrors(runtime_status);
	runtime_status = hipHostFree(hostbgrright);
	checkCudaErrors(runtime_status);
	runtime_status = hipHostFree(hostgrayleft);
	checkCudaErrors(runtime_status);
	runtime_status = hipHostFree(hostgrayright);
	checkCudaErrors(runtime_status);
	runtime_status = hipHostFree(hostgradleft);
	checkCudaErrors(runtime_status);
	runtime_status = hipHostFree(hostgradright);
	checkCudaErrors(runtime_status);
	runtime_status = hipHostFree(hostdispleft);
	checkCudaErrors(runtime_status);
	runtime_status = hipHostFree(hostdispright);
	checkCudaErrors(runtime_status);
	runtime_status = hipHostFree(hostdispplaneleft);
	checkCudaErrors(runtime_status);
	runtime_status = hipHostFree(hostdispplaneright);
	checkCudaErrors(runtime_status);
	runtime_status = hipHostFree(hostcostaggrleft);
	checkCudaErrors(runtime_status);
	runtime_status = hipHostFree(hostcostaggrright);
	checkCudaErrors(runtime_status);
	runtime_status = hipHostFree(host_aggr_offset);
	checkCudaErrors(runtime_status);
	runtime_status = hipHostFree(host_aggr3_offset);
	checkCudaErrors(runtime_status);


	runtime_status = hipFree(devbgrleft);
	checkCudaErrors(runtime_status);
	runtime_status = hipFree(devbgrright);
	checkCudaErrors(runtime_status);
	runtime_status = hipFree(devgrayleft);
	checkCudaErrors(runtime_status);
	runtime_status = hipFree(devgrayright);
	checkCudaErrors(runtime_status);
	runtime_status = hipFree(devgradleft);
	checkCudaErrors(runtime_status);
	runtime_status = hipFree(devgradright);
	checkCudaErrors(runtime_status);

	runtime_status = hipFree(devdispleft);
	checkCudaErrors(runtime_status);
	runtime_status = hipFree(devdispright);
	checkCudaErrors(runtime_status);
	runtime_status = hipFree(devdispplaneleft);
	checkCudaErrors(runtime_status);
	runtime_status = hipFree(devdispplaneright);
	checkCudaErrors(runtime_status);


	runtime_status = hipFree(devcostaggrleft);
	checkCudaErrors(runtime_status);
	runtime_status = hipFree(devcostaggrright);
	checkCudaErrors(runtime_status);


	return 0;
}
