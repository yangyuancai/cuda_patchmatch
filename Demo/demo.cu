#include "hip/hip_runtime.h"
#include <stdio.h>
#include "headfile.h"
#include "kernels.h"

#define DEBUG 1

#ifdef DEBUG
	#include "opencv2/opencv.hpp"
#endif // DEBUG


#define checkCudaErrors(val) check((val), #val, __FILE__, __LINE__)
template <typename T>
void check(T result, char const* const func, const char* const file, int const line) {
	if (result) {
		fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\" \n", file, line,
			static_cast<unsigned int>(result), hipGetErrorName(result), func);
		exit(EXIT_FAILURE);
	}
}

int main()
{
	hipError_t cuda_runtime_status = hipSuccess;
	hiprandStatus_t cuda_rand_status = HIPRAND_STATUS_SUCCESS;
	ImageInfo bgrInfo(450u, 375u, Uchar3Img);
	ImageInfo grayInfo(450u, 375u, UcharImg);

	hiprandGenerator_t generator;
	hiprandStatus_t rand_status;
	hipError_t runtime_status;
	rand_status = hiprandCreateGenerator(&generator, HIPRAND_RNG_QUASI_SCRAMBLED_SOBOL32);
	if (HIPRAND_STATUS_SUCCESS != rand_status)
		return rand_status;

	// memory allocate
	uchar *hostbgrleft = NULL;
	uchar *hostbgrright = NULL;

	float* hostgrayleft = NULL;
	float* hostgrayright = NULL;
	float3* hostgradleft = NULL;
	float3* hostgradright = NULL;

	float *hostdispleft = NULL;
	float *hostdispright = NULL;
	float3 *hostdispplaneleft = NULL;
	float3 *hostdispplaneright = NULL;

	uchar3* devbgrleft = NULL;
	uchar3* devbgrright = NULL;
	float* devgrayleft = NULL;
	float* devgrayright = NULL;
	float3* devgradleft = NULL;
	float3* devgradright = NULL;

	float *devcostleft = NULL;
	float *devcostright = NULL;
	float *devdispleft = NULL;
	float *devdispright = NULL;
	float3 *devdispplaneleft = NULL;
	float3 *devdispplaneright = NULL;

	cuda_runtime_status = hipHostMalloc((void**)&hostbgrleft, sizeof(uchar) * grayInfo.imgsize * 3);
	checkCudaErrors(cuda_runtime_status);
	cuda_runtime_status = hipHostMalloc((void**)&hostbgrright, sizeof(uchar) * grayInfo.imgsize * 3);
	checkCudaErrors(cuda_runtime_status);


	cuda_runtime_status = hipHostMalloc((void**)&hostgrayleft, sizeof(float)*grayInfo.imgsize);
	checkCudaErrors(cuda_runtime_status);
	cuda_runtime_status = hipHostMalloc((void**)&hostgrayright, sizeof(float) * grayInfo.imgsize);
	checkCudaErrors(cuda_runtime_status);
	cuda_runtime_status = hipHostMalloc((void**)&hostgradleft, sizeof(float3) * grayInfo.imgsize);
	checkCudaErrors(cuda_runtime_status);
	cuda_runtime_status = hipHostMalloc((void**)&hostgradright, sizeof(float3) * grayInfo.imgsize);
	checkCudaErrors(cuda_runtime_status);

	cuda_runtime_status = hipHostMalloc((void**)&hostdispleft, sizeof(float) * grayInfo.imgsize);
	checkCudaErrors(cuda_runtime_status);
	cuda_runtime_status = hipHostMalloc((void**)&hostdispright, sizeof(float) * grayInfo.imgsize);
	checkCudaErrors(cuda_runtime_status);
	cuda_runtime_status = hipHostMalloc((void**)&hostdispplaneleft, sizeof(float3) * grayInfo.imgsize);
	checkCudaErrors(cuda_runtime_status);
	cuda_runtime_status = hipHostMalloc((void**)&hostdispplaneright, sizeof(float3) * grayInfo.imgsize);
	checkCudaErrors(cuda_runtime_status);


	cuda_runtime_status = hipMalloc((void**)&devgrayleft, sizeof(float) * grayInfo.imgsize);
	checkCudaErrors(cuda_runtime_status);
	cuda_runtime_status = hipMalloc((void**)&devgrayright, sizeof(float) * grayInfo.imgsize);
	checkCudaErrors(cuda_runtime_status);
	cuda_runtime_status = hipMalloc((void**)&devbgrleft, sizeof(float) * grayInfo.imgsize);
	checkCudaErrors(cuda_runtime_status);
	cuda_runtime_status = hipMalloc((void**)&devbgrright, sizeof(float) * grayInfo.imgsize);
	checkCudaErrors(cuda_runtime_status);
	cuda_runtime_status = hipMalloc((void**)&devgradleft, sizeof(float3) * grayInfo.imgsize);
	checkCudaErrors(cuda_runtime_status);
	cuda_runtime_status = hipMalloc((void**)&devgradright, sizeof(float3) * grayInfo.imgsize);
	checkCudaErrors(cuda_runtime_status);
	cuda_runtime_status = hipMalloc((void**)&devcostleft, sizeof(float) * grayInfo.imgsize);
	checkCudaErrors(cuda_runtime_status);
	cuda_runtime_status = hipMalloc((void**)&devcostright, sizeof(float) * grayInfo.imgsize);
	checkCudaErrors(cuda_runtime_status);
	cuda_runtime_status = hipMalloc((void**)&devdispleft, sizeof(float) * grayInfo.imgsize);
	checkCudaErrors(cuda_runtime_status);
	cuda_runtime_status = hipMalloc((void**)&devdispright, sizeof(float) * grayInfo.imgsize);
	checkCudaErrors(cuda_runtime_status);
	cuda_runtime_status = hipMalloc((void**)&devdispplaneleft, sizeof(float3) * grayInfo.imgsize);
	checkCudaErrors(cuda_runtime_status);
	cuda_runtime_status = hipMalloc((void**)&devdispplaneright, sizeof(float3) * grayInfo.imgsize);
	checkCudaErrors(cuda_runtime_status);

	cuda_runtime_status = hipMemset(devdispleft, 0, sizeof(float) * grayInfo.imgsize);
	checkCudaErrors(cuda_runtime_status);
	cuda_runtime_status = hipMemset(devdispright, 0, sizeof(float) * grayInfo.imgsize);
	checkCudaErrors(cuda_runtime_status);
	cuda_runtime_status = hipMemset(devdispplaneleft, 0, sizeof(float3) * grayInfo.imgsize);
	checkCudaErrors(cuda_runtime_status);
	cuda_runtime_status = hipMemset(devdispplaneright, 0, sizeof(float3) * grayInfo.imgsize);
	checkCudaErrors(cuda_runtime_status);
	std::cout << sizeof(float3) << std::endl;


	// init device disparity plane
	rand_status = hiprandGenerateNormal(generator, devdispleft, grayInfo.imgsize, 0.0, 1.0);
	checkCudaErrors(hipError_t(cuda_rand_status));
	rand_status = hiprandGenerateNormal(generator, devdispright, grayInfo.imgsize, 0.0, 1.0);
	checkCudaErrors(hipError_t(cuda_rand_status));
	
	rand_status = hiprandGenerateNormal(generator, (float*)(void*)devdispplaneleft, grayInfo.imgsize*3, 0.0, 1.0);
	checkCudaErrors(hipError_t(cuda_rand_status));
	rand_status = hiprandGenerateNormal(generator, (float*)(void*)devdispplaneright,grayInfo.imgsize*3, 0.0, 1.0);
	checkCudaErrors(hipError_t(cuda_rand_status));



	#ifdef DEBUG

	cuda_runtime_status = hipMemcpy(hostdispleft, devdispleft, sizeof(float) * grayInfo.imgsize, hipMemcpyDeviceToHost);
	checkCudaErrors(cuda_runtime_status);
	cuda_runtime_status = hipMemcpy(hostdispright, devdispright, sizeof(float) * grayInfo.imgsize, hipMemcpyDeviceToHost);
	checkCudaErrors(cuda_runtime_status);
	cuda_runtime_status = hipMemcpy(hostdispplaneleft, devdispplaneleft, sizeof(float3) * grayInfo.imgsize, hipMemcpyDeviceToHost);
	checkCudaErrors(cuda_runtime_status);
	cuda_runtime_status = hipMemcpy(hostdispplaneright, devdispplaneright, sizeof(float3) * grayInfo.imgsize, hipMemcpyDeviceToHost);
	checkCudaErrors(cuda_runtime_status);
	hipDeviceSynchronize();

	cv::Mat hostdispleft_mat = cv::Mat(grayInfo.height, grayInfo.width, CV_32FC1, hostdispleft);
	cv::Mat hostdispright_mat = cv::Mat(grayInfo.height, grayInfo.width, CV_32FC1, hostdispright);
	cv::Mat hostdispplaneleft_mat = cv::Mat(grayInfo.height, grayInfo.width, CV_32FC3, hostdispplaneleft);
	cv::Mat hostdispplaneright_mat = cv::Mat(grayInfo.height, grayInfo.width, CV_32FC3, hostdispplaneright);
	#endif // DEBUG


	//load image from host memory
	FILE *fptr = NULL;
	fptr = fopen("../../data/Cone/left.raw", "rb");
	fread(hostbgrleft, sizeof(uchar), bgrInfo.width * bgrInfo.height * 3, fptr);
	fclose(fptr);
	fptr = fopen("../../data/Cone/right.raw", "rb");
	fread(hostbgrright, sizeof(uchar), bgrInfo.width * bgrInfo.height * 3, fptr);
	fclose(fptr);

	#ifdef DEBUG
		cv::Mat host_bgr_left_mat = cv::Mat(grayInfo.height, grayInfo.width, CV_8UC3, hostbgrleft);
		cv::Mat host_bgr_right_mat = cv::Mat(grayInfo.height, grayInfo.width, CV_8UC3, hostbgrright);
	#endif //DEBUG


	//Copy image to device
	cuda_runtime_status = hipMemcpy(devbgrleft, hostbgrleft, sizeof(char3) * grayInfo.imgsize, hipMemcpyHostToDevice);
	checkCudaErrors(cuda_runtime_status);
	cuda_runtime_status = hipMemcpy(devbgrright, hostbgrright, sizeof(char3) * grayInfo.imgsize, hipMemcpyHostToDevice);
	checkCudaErrors(cuda_runtime_status);
	
	//convert bgr image to gray and grad image
	dim3 threadsperblock(32u, 32u);
	dim3 blockpergrid(15u, 12u);
	Bgr2Gray(devgrayleft, devbgrleft,grayInfo.width, grayInfo.height, blockpergrid, threadsperblock);
	Bgr2Gray(devgrayright, devbgrright,grayInfo.width, grayInfo.height, blockpergrid, threadsperblock);
	Gray2Sobel(devgradleft,devgrayleft,  grayInfo.width, grayInfo.height, blockpergrid, threadsperblock);
	Gray2Sobel(devgradright,devgrayright, grayInfo.width, grayInfo.height, blockpergrid, threadsperblock);
	
	//random initialiation 

	//load image to host
#ifdef DEBUG
	cuda_runtime_status = hipMemcpy(hostgrayleft, devgrayleft, sizeof(float) * grayInfo.imgsize, hipMemcpyDeviceToHost);
	checkCudaErrors(cuda_runtime_status);
	cuda_runtime_status = hipMemcpy(hostgrayright, devgrayright, sizeof(float) * grayInfo.imgsize, hipMemcpyDeviceToHost);
	checkCudaErrors(cuda_runtime_status);
	cuda_runtime_status = hipMemcpy(hostgradleft, devgradleft, sizeof(float3) * grayInfo.imgsize, hipMemcpyDeviceToHost);
	checkCudaErrors(cuda_runtime_status);
	cuda_runtime_status = hipMemcpy(hostgradright, devgradright, sizeof(float3) * grayInfo.imgsize, hipMemcpyDeviceToHost);
	checkCudaErrors(cuda_runtime_status);

	hipDeviceSynchronize();

	cv::Mat gray_left_mat = cv::Mat(grayInfo.height, grayInfo.width, CV_32FC1, hostgrayleft);
	cv::Mat grad_left_mat = cv::Mat(grayInfo.height, grayInfo.width, CV_32FC3, hostgradleft);
	cv::Mat gray_right_mat = cv::Mat(grayInfo.height, grayInfo.width, CV_32FC1, hostgrayright);
	cv::Mat grad_right_mat = cv::Mat(grayInfo.height, grayInfo.width, CV_32FC3, hostgradright);
#endif //DEBUG



	hipFree(hostbgrleft);
	hipFree(hostbgrright);
	hipFree(hostgrayleft);
	hipFree(hostgrayright);
	hipFree(hostgradleft);
	hipFree(hostgradright);
	hipFree(hostdispleft);
	hipFree(hostdispright);
	hipFree(hostdispplaneleft);
	hipFree(hostdispplaneright);
			  
	hipFree(devbgrleft);
	hipFree(devbgrright);
	hipFree(devgrayleft);
	hipFree(devgrayright);
	hipFree(devgradleft);
	hipFree(devgradright);
	hipFree(devcostleft);
	hipFree(devcostright);
	hipFree(devdispleft);
	hipFree(devdispright);
	hipFree(devdispplaneleft);
	hipFree(devdispplaneright);

	return 0;
}














//cv::Mat hostbgrleft = cv::imread("../../data/Cone/left.png", 1);
//cv::Mat hostbgrright = cv::imread("../../data/Cone/right.png", 1);
//FILE *fptr = NULL;
//fptr = fopen("../../data/Cone/left.raw", "wb");
//fwrite(hostbgrleft.data,sizeof(uchar),450*375*3, fptr);
//fclose(fptr);
//FILE *fptr2 = NULL;
//fptr2 = fopen("../../data/Cone/right.raw", "wb");
//fwrite(hostbgrright.data, sizeof(uchar), 450 * 375*3, fptr2);
//fclose(fptr);